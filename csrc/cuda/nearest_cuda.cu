#include "hip/hip_runtime.h"
#include "nearest_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024

template <typename scalar_t>
__global__ void nearest_kernel(const scalar_t *x, const scalar_t *y,
                               const int64_t *ptr_x, const int64_t *ptr_y,
                               int64_t *out, int64_t batch_size, int64_t dim) {

  const int64_t thread_idx = threadIdx.x;
  const int64_t n_x = blockIdx.x;

  int64_t batch_idx;
  for (int64_t b = 0; b < batch_size; b++) {
    if (n_x >= ptr_x[b] && n_x < ptr_x[b + 1]) {
      batch_idx = b;
      break;
    }
  }

  const int64_t y_start_idx = ptr_y[batch_idx];
  const int64_t y_end_idx = ptr_y[batch_idx + 1];

  __shared__ scalar_t best_dist[THREADS];
  __shared__ int64_t best_dist_idx[THREADS];

  scalar_t best = 1e38;
  int64_t best_idx = 0;
  for (int64_t n_y = y_start_idx + thread_idx; n_y < y_end_idx;
       n_y += THREADS) {
    scalar_t dist = 0;
    for (int64_t d = 0; d < dim; d++) {
      dist += (x[n_x * dim + d] - y[n_y * dim + d]) *
              (x[n_x * dim + d] - y[n_y * dim + d]);
    }

    if (dist < best) {
      best = dist;
      best_idx = n_y;
    }
  }

  best_dist[thread_idx] = best;
  best_dist_idx[thread_idx] = best_idx;

  for (int64_t u = 0; (1 << u) < THREADS; u++) {
    __syncthreads();
    if (thread_idx < (THREADS >> (u + 1))) {
      int64_t idx_1 = (thread_idx * 2) << u;
      int64_t idx_2 = (thread_idx * 2 + 1) << u;
      if (best_dist[idx_1] > best_dist[idx_2]) {
        best_dist[idx_1] = best_dist[idx_2];
        best_dist_idx[idx_1] = best_dist_idx[idx_2];
      }
    }
  }

  __syncthreads();
  if (thread_idx == 0) {
    out[n_x] = best_dist_idx[0];
  }
}

torch::Tensor nearest_cuda(torch::Tensor x, torch::Tensor y,
                           torch::Tensor ptr_x, torch::Tensor ptr_y) {
  CHECK_CUDA(x);
  CHECK_CUDA(y);
  CHECK_CUDA(ptr_x);
  CHECK_CUDA(ptr_y);
  hipSetDevice(x.get_device());

  x = x.view({x.size(0), -1}).contiguous();
  y = y.view({y.size(0), -1}).contiguous();

  auto out = torch::empty({x.size(0)}, ptr_x.options());

  auto stream = at::cuda::getCurrentCUDAStream();
  auto scalar_type = x.scalar_type();
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, scalar_type, "_", [&] {
    nearest_kernel<scalar_t><<<x.size(0), THREADS, 0, stream>>>(
        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(),
        ptr_x.data_ptr<int64_t>(), ptr_y.data_ptr<int64_t>(),
        out.data_ptr<int64_t>(), ptr_x.size(0) - 1, x.size(1));
  });

  return out;
}
