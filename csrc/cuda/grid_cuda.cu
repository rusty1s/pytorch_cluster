#include "hip/hip_runtime.h"
#include "grid_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t>
__global__ void grid_kernel(const scalar_t *pos, const scalar_t *size,
                            const scalar_t *start, const scalar_t *end,
                            int64_t *out, int64_t D, int64_t numel) {
  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    int64_t c = 0, k = 1;
    for (int64_t d = 0; d < D; d++) {
      scalar_t p = pos[thread_idx * D + d] - start[d];
      c += (int64_t)(p / size[d]) * k;
      k *= (int64_t)((end[d] - start[d]) / size[d]) + 1;
    }
    out[thread_idx] = c;
  }
}

torch::Tensor grid_cuda(torch::Tensor pos, torch::Tensor size,
                        torch::optional<torch::Tensor> optional_start,
                        torch::optional<torch::Tensor> optional_end) {
  CHECK_CUDA(pos);
  CHECK_CUDA(size);
  hipSetDevice(pos.get_device());

  if (optional_start.has_value())
    CHECK_CUDA(optional_start.value());
  if (optional_start.has_value())
    CHECK_CUDA(optional_start.value());

  pos = pos.view({pos.size(0), -1}).contiguous();
  size = size.contiguous();

  CHECK_INPUT(size.numel() == pos.size(1));

  if (!optional_start.has_value())
    optional_start = std::get<0>(pos.min(0));
  else {
    optional_start = optional_start.value().contiguous();
    CHECK_INPUT(optional_start.value().numel() == pos.size(1));
  }

  if (!optional_end.has_value())
    optional_end = std::get<0>(pos.max(0));
  else {
    optional_start = optional_start.value().contiguous();
    CHECK_INPUT(optional_start.value().numel() == pos.size(1));
  }

  auto start = optional_start.value();
  auto end = optional_end.value();

  auto out = torch::empty(pos.size(0), pos.options().dtype(torch::kLong));

  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, pos.scalar_type(), "_", [&] {
    grid_kernel<scalar_t><<<BLOCKS(out.numel()), THREADS, 0, stream>>>(
        pos.data_ptr<scalar_t>(), size.data_ptr<scalar_t>(),
        start.data_ptr<scalar_t>(), end.data_ptr<scalar_t>(),
        out.data_ptr<int64_t>(), pos.size(1), out.numel());
  });

  return out;
}
