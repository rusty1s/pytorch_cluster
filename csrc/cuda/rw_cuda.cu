#include "hip/hip_runtime.h"
#include "rw_cuda.h"

#include <ATen/cuda/HIPContext.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

__global__ void uniform_sampling_kernel(const int64_t *rowptr,
                                        const int64_t *col,
                                        const int64_t *start, const float *rand,
                                        int64_t *n_out, int64_t *e_out,
                                        const int64_t walk_length,
                                        const int64_t numel) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    int64_t n_cur = start[thread_idx], e_cur, row_start, row_end, rnd;

    n_out[thread_idx] = n_cur;

    for (int64_t l = 0; l < walk_length; l++) {
      row_start = rowptr[n_cur], row_end = rowptr[n_cur + 1];
      if (row_end - row_start == 0) {
        e_cur = -1;
      } else {
        rnd = int64_t(rand[l * numel + thread_idx] * (row_end - row_start));
        e_cur = row_start + rnd;
        n_cur = col[e_cur];
      }
      n_out[(l + 1) * numel + thread_idx] = n_cur;
      e_out[l * numel + thread_idx] = e_cur;
    }
  }
}


__global__ void cdf_kernel(const int64_t *rowptr, const float_t *edge_weight,
		           float_t *edge_weight_cdf, int64_t numel) {
  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel - 1) {
    int64_t row_start = rowptr[thread_idx], row_end = rowptr[thread_idx + 1];

    float_t sum = 0.0;

    for(int64_t i = row_start; i < row_end; i++) {
      sum += edge_weight[i];
    }

    float_t acc = 0.0;

    for(int64_t i = row_start; i < row_end; i++) {
      acc += edge_weight[i] / sum;
      edge_weight_cdf[i] = acc;
    }
  }
}

__device__ void get_offset(const float_t *edge_weight, int64_t start, int64_t end,
                           float_t value, int64_t *position_out) {
  int64_t original_start = start;

  while (start < end) {
    const int64_t mid = start + ((end - start) >> 1);
    const float_t mid_val = edge_weight[mid];
    if (!(mid_val >= value)) {
      start = mid + 1;
    }
    else {
      end = mid;
    }
  }

  *position_out = start - original_start;
}

__global__ void
rejection_sampling_weighted_kernel(unsigned int seed, const int64_t *rowptr,
                                   const int64_t *col, const float_t *edge_weight_cdf,
                                   const int64_t *start, int64_t *n_out,
                                   int64_t *e_out, const int64_t walk_length,
                                   const int64_t numel, const double p,
                                   const double q) {

  hiprandState_t state;
  hiprand_init(seed, 0, 0, &state);

  double max_prob = fmax(fmax(1. / p, 1.), 1. / q);
  double prob_0 = 1. / p / max_prob;
  double prob_1 = 1. / max_prob;
  double prob_2 = 1. / q / max_prob;

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    int64_t t = start[thread_idx], v, x, e_cur, row_start, row_end, offset;

    n_out[thread_idx] = t;

    row_start = rowptr[t], row_end = rowptr[t + 1];

    if (row_end - row_start == 0) {
      e_cur = -1;
      v = t;
    } else {
      get_offset(edge_weight_cdf, row_start, row_end, hiprand_uniform(&state), &offset);
      e_cur = row_start + offset;
      v = col[e_cur];
    }

    n_out[numel + thread_idx] = v;
    e_out[thread_idx] = e_cur;

    for (int64_t l = 1; l < walk_length; l++) {
      row_start = rowptr[v], row_end = rowptr[v + 1];

      if (row_end - row_start == 0) {
        e_cur = -1;
        x = v;
      } else if (row_end - row_start == 1) {
        e_cur = row_start;
        x = col[e_cur];
      } else {
        if (p == 1. && q == 1.) {
          get_offset(edge_weight_cdf, row_start, row_end, hiprand_uniform(&state), &offset);
          e_cur = row_start + offset;
          x = col[e_cur];
        }
        else {
          while (true) {
            get_offset(edge_weight_cdf, row_start, row_end, hiprand_uniform(&state), &offset);
            e_cur = row_start + offset;
            x = col[e_cur];

            double r = hiprand_uniform(&state); // (0, 1]

            if (x == t && r < prob_0)
              break;

            bool is_neighbor = false;
            row_start = rowptr[x], row_end = rowptr[x + 1];
            for (int64_t i = row_start; i < row_end; i++) {
              if (col[i] == t) {
                is_neighbor = true;
                break;
              }
            }

            if (is_neighbor && r < prob_1)
              break;
            else if (r < prob_2)
              break;
          }
        }
      }

      n_out[(l + 1) * numel + thread_idx] = x;
      e_out[l * numel + thread_idx] = e_cur;
      t = v;
      v = x;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor>
random_walk_weighted_cuda(torch::Tensor rowptr, torch::Tensor col,
                          torch::Tensor edge_weight, torch::Tensor start,
                          int64_t walk_length, double p, double q) {
  CHECK_CUDA(rowptr);
  CHECK_CUDA(col);
  CHECK_CUDA(edge_weight);
  CHECK_CUDA(start);
  hipSetDevice(rowptr.get_device());

  CHECK_INPUT(rowptr.dim() == 1);
  CHECK_INPUT(col.dim() == 1);
  CHECK_INPUT(edge_weight.dim() == 1);
  CHECK_INPUT(start.dim() == 1);

  auto n_out = torch::empty({walk_length + 1, start.size(0)}, start.options());
  auto e_out = torch::empty({walk_length, start.size(0)}, start.options());

  auto stream = at::cuda::getCurrentCUDAStream();

  auto edge_weight_cdf = torch::empty({edge_weight.size(0)}, edge_weight.options());

  cdf_kernel<<<BLOCKS(rowptr.numel()), THREADS, 0, stream>>>(
      rowptr.data_ptr<int64_t>(), edge_weight.data_ptr<float_t>(),
      edge_weight_cdf.data_ptr<float_t>(), rowptr.numel());

  rejection_sampling_weighted_kernel<<<BLOCKS(start.numel()), THREADS, 0, stream>>>(
      time(NULL), rowptr.data_ptr<int64_t>(), col.data_ptr<int64_t>(),
      edge_weight_cdf.data_ptr<float_t>(), start.data_ptr<int64_t>(),
      n_out.data_ptr<int64_t>(), e_out.data_ptr<int64_t>(),
      walk_length, start.numel(), p, q);

  return std::make_tuple(n_out.t().contiguous(), e_out.t().contiguous());
}

__global__ void
rejection_sampling_kernel(unsigned int seed, const int64_t *rowptr,
                          const int64_t *col, const int64_t *start,
                          int64_t *n_out, int64_t *e_out,
                          const int64_t walk_length, const int64_t numel,
                          const double p, const double q) {

  hiprandState_t state;
  hiprand_init(seed, 0, 0, &state);

  double max_prob = fmax(fmax(1. / p, 1.), 1. / q);
  double prob_0 = 1. / p / max_prob;
  double prob_1 = 1. / max_prob;
  double prob_2 = 1. / q / max_prob;

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    int64_t t = start[thread_idx], v, x, e_cur, row_start, row_end;

    n_out[thread_idx] = t;

    row_start = rowptr[t], row_end = rowptr[t + 1];
    if (row_end - row_start == 0) {
      e_cur = -1;
      v = t;
    } else {
      e_cur = row_start + (hiprand(&state) % (row_end - row_start));
      v = col[e_cur];
    }

    n_out[numel + thread_idx] = v;
    e_out[thread_idx] = e_cur;

    for (int64_t l = 1; l < walk_length; l++) {
      row_start = rowptr[v], row_end = rowptr[v + 1];

      if (row_end - row_start == 0) {
        e_cur = -1;
        x = v;
      } else if (row_end - row_start == 1) {
        e_cur = row_start;
        x = col[e_cur];
      } else {
        while (true) {
          e_cur = row_start + (hiprand(&state) % (row_end - row_start));
          x = col[e_cur];

          double r = hiprand_uniform(&state); // (0, 1]

          if (x == t && r < prob_0)
            break;

          bool is_neighbor = false;
          row_start = rowptr[x], row_end = rowptr[x + 1];
          for (int64_t i = row_start; i < row_end; i++) {
            if (col[i] == t) {
              is_neighbor = true;
              break;
            }
          }

          if (is_neighbor && r < prob_1)
            break;
          else if (r < prob_2)
            break;
        }
      }

      n_out[(l + 1) * numel + thread_idx] = x;
      e_out[l * numel + thread_idx] = e_cur;
      t = v;
      v = x;
    }
  }
}

std::tuple<torch::Tensor, torch::Tensor>
random_walk_cuda(torch::Tensor rowptr, torch::Tensor col, torch::Tensor start,
                 int64_t walk_length, double p, double q) {
  CHECK_CUDA(rowptr);
  CHECK_CUDA(col);
  CHECK_CUDA(start);
  hipSetDevice(rowptr.get_device());

  CHECK_INPUT(rowptr.dim() == 1);
  CHECK_INPUT(col.dim() == 1);
  CHECK_INPUT(start.dim() == 1);

  auto n_out = torch::empty({walk_length + 1, start.size(0)}, start.options());
  auto e_out = torch::empty({walk_length, start.size(0)}, start.options());

  auto stream = at::cuda::getCurrentCUDAStream();

  if (p == 1. && q == 1.) {
    auto rand = torch::rand({start.size(0), walk_length},
                            start.options().dtype(torch::kFloat));

    uniform_sampling_kernel<<<BLOCKS(start.numel()), THREADS, 0, stream>>>(
        rowptr.data_ptr<int64_t>(), col.data_ptr<int64_t>(),
        start.data_ptr<int64_t>(), rand.data_ptr<float>(),
        n_out.data_ptr<int64_t>(), e_out.data_ptr<int64_t>(), walk_length,
        start.numel());
  } else {
    rejection_sampling_kernel<<<BLOCKS(start.numel()), THREADS, 0, stream>>>(
        time(NULL), rowptr.data_ptr<int64_t>(), col.data_ptr<int64_t>(),
        start.data_ptr<int64_t>(), n_out.data_ptr<int64_t>(),
        e_out.data_ptr<int64_t>(), walk_length, start.numel(), p, q);
  }

  return std::make_tuple(n_out.t().contiguous(), e_out.t().contiguous());
}
