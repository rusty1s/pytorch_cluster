#include "hip/hip_runtime.h"
#include "radius_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 256

template <typename scalar_t>
__global__ void
radius_kernel(const scalar_t *__restrict__ x, const scalar_t *__restrict__ y,
              const int64_t *__restrict__ ptr_x,
              const int64_t *__restrict__ ptr_y, int64_t *__restrict__ row,
              int64_t *__restrict__ col, const scalar_t r, const int64_t n,
              const int64_t m, const int64_t dim, const int64_t num_examples,
              const int64_t max_num_neighbors) {

  const int64_t n_y = blockIdx.x * blockDim.x + threadIdx.x;
  if (n_y >= m)
    return;

  int64_t count = 0;
  const int64_t example_idx = get_example_idx(n_y, ptr_y, num_examples);

  for (int64_t n_x = ptr_x[example_idx]; n_x < ptr_x[example_idx + 1]; n_x++) {
    scalar_t dist = 0;
    for (int64_t d = 0; d < dim; d++) {
      dist += (x[n_x * dim + d] - y[n_y * dim + d]) *
              (x[n_x * dim + d] - y[n_y * dim + d]);
    }

    if (dist < r) {
      row[n_y * max_num_neighbors + count] = n_y;
      col[n_y * max_num_neighbors + count] = n_x;
      count++;
    }

    if (count >= max_num_neighbors)
      break;
  }
}

torch::Tensor radius_cuda(const torch::Tensor x, const torch::Tensor y,
                          torch::optional<torch::Tensor> ptr_x,
                          torch::optional<torch::Tensor> ptr_y, const double r,
                          const int64_t max_num_neighbors) {
  CHECK_CUDA(x);
  CHECK_CONTIGUOUS(x);
  CHECK_INPUT(x.dim() == 2);
  CHECK_CUDA(y);
  CHECK_CONTIGUOUS(y);
  CHECK_INPUT(y.dim() == 2);
  CHECK_INPUT(x.size(1) == y.size(1));

  hipSetDevice(x.get_device());

  if (ptr_x.has_value()) {
    CHECK_CUDA(ptr_x.value());
    CHECK_INPUT(ptr_x.value().dim() == 1);
  } else
    ptr_x = torch::arange(0, x.size(0) + 1, x.size(0),
                          x.options().dtype(torch::kLong));

  if (ptr_y.has_value()) {
    CHECK_CUDA(ptr_y.value());
    CHECK_INPUT(ptr_y.value().dim() == 1);
  } else
    ptr_y = torch::arange(0, y.size(0) + 1, y.size(0),
                          y.options().dtype(torch::kLong));

  CHECK_INPUT(ptr_x.value().numel() == ptr_y.value().numel());

  hipSetDevice(x.get_device());

  auto row =
      torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.value().options());
  auto col =
      torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.value().options());

  dim3 BLOCKS((y.size(0) + THREADS - 1) / THREADS);

  auto stream = at::cuda::getCurrentCUDAStream();
  auto scalar_type = x.scalar_type();
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, scalar_type, "_", [&] {
    radius_kernel<scalar_t><<<BLOCKS, THREADS, 0, stream>>>(
        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(),
        ptr_x.value().data_ptr<int64_t>(), ptr_y.value().data_ptr<int64_t>(),
        row.data_ptr<int64_t>(), col.data_ptr<int64_t>(), r * r, x.size(0),
        y.size(0), x.size(1), ptr_x.value().numel() - 1, max_num_neighbors);
  });

  auto mask = row != -1;
  return torch::stack({row.masked_select(mask), col.masked_select(mask)}, 0);
}
